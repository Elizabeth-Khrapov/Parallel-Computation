#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "general.h"
#include "cudaFunctions.h"
#include "ompFunctions.h"
__constant__ const char CONSERVATIVE[9][4] = { { 'N', 'D', 'E', 'Q' },
		{ 'N', 'E', 'Q', 'K' }, { 'S', 'T', 'A', '^' }, { 'M', 'I', 'L', 'V' },
		{ 'Q', 'H', 'R', 'K' }, { 'N', 'H', 'Q', 'K' }, { 'F', 'Y', 'W', '^' },
		{ 'H', 'Y', '^', '^' }, { 'M', 'I', 'L', 'F' } };
__constant__ const char SEMI_CONSERVATIVE[11][6] = { { 'S', 'A', 'G', '^', '^', '^' }, { 'A',
		'T', 'V', '^', '^', '^' }, { 'C', 'S', 'A', '^', '^', '^' }, { 'S', 'G',
		'N', 'D', '^', '^' }, { 'S', 'T', 'P', 'A', '^', '^' }, { 'S', 'T', 'N',
		'K', '^', '^' }, { 'N', 'E', 'Q', 'H', 'R', 'K' }, { 'N', 'D', 'E', 'Q',
		'H', 'K' }, { 'S', 'N', 'D', 'E', 'Q', 'K' }, { 'H', 'F', 'Y', '^', '^',
		'^' }, { 'F', 'V', 'L', 'I', 'M', '^' } };
__constant__  int ROW_CONSERVATIVE = 9;
__constant__  int COL_CONSERVATIVE = 4;
__constant__  int ROW_SEMI_CONSERVATIVE = 11;
__constant__  int COL_SEMI_CONSERVATIVE = 6;
__device__ int checkColonSign(char a, char b) {
	int aFound;
	int bFound;
	int i, j;
	for (i = 0; i < ROW_CONSERVATIVE; i++) {
		aFound = 0;
		bFound = 0;
		for (j = 0; j < COL_CONSERVATIVE; j++) {
			if (!aFound && a == CONSERVATIVE[i][j]) {
				aFound = 1;
			}
			if (!bFound && b == CONSERVATIVE[i][j]) {
				bFound = 1;
			}
			if (aFound && bFound) {
				return 1;
			}
		}
	}
	return 0;
}
__device__ int checkPointSign(char a, char b) {
	int aFound;
	int bFound;
	int i, j;
	for (i = 0; i < ROW_SEMI_CONSERVATIVE; i++) {
		aFound = 0;
		bFound = 0;
		for (j = 0; j < COL_SEMI_CONSERVATIVE; j++) {
			if (!aFound && a == SEMI_CONSERVATIVE[i][j]) {
				aFound = 1;
			}
			if (!bFound && b == SEMI_CONSERVATIVE[i][j]) {
				bFound = 1;
			}
			if (aFound && bFound) {
				return 1;
			}
		}
	}
	return 0;
}

__global__ void getSignOfTwoLettersCompare(char* seq1_cuda, char* seq2_cuda, char* signs_cuda, int length, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < length) {
		signs_cuda[i] = ' ';
		if (seq2_cuda[i] == '-') {
			return;
		}
		if (seq1_cuda[i+n] == seq2_cuda[i]) {
			signs_cuda[i] = '*';
		}
		else if (checkColonSign(seq1_cuda[i+n], seq2_cuda[i])) {
			signs_cuda[i] = ':';
		}
		else if (checkPointSign(seq1_cuda[i+n], seq2_cuda[i])) {
			signs_cuda[i] = '.';
		}
	}
}

BestScore cudaFunction(char *seq1, char *seq2, int sizeOfSeq1, int sizeOfSeq2,int k ,double *w) {
        
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate memory on GPU to copy the data from the host
    char* seq1_cuda;
char* seq2_cuda;
char* signs_cuda;
    err = hipMalloc((void **)&seq1_cuda, sizeOfSeq1);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
 }
    err = hipMalloc((void **)&seq2_cuda, sizeOfSeq2);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
 }
    err = hipMalloc((void **)&signs_cuda, sizeOfSeq2);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
 }
    // Copy data from host to the GPU memory
    err = hipMemcpy(seq1_cuda,seq1, sizeOfSeq1,hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(seq2_cuda,seq2, sizeOfSeq2,hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    int threadsPerBlock = (sizeOfSeq2 > 1024 ? 1024 : sizeOfSeq2);
    int blocksPerGrid =(sizeOfSeq2 / threadsPerBlock) +(sizeOfSeq2 % threadsPerBlock != 0);

/* Loop Start */
int i;
	BestScore best, tempBest;
	best.score = -INFINITY;
   	char* stringSigns;
	for (i = 0; i < sizeOfSeq1 - sizeOfSeq2 + 1; i++) {
   	getSignOfTwoLettersCompare<<<blocksPerGrid, threadsPerBlock>>>(seq1_cuda,seq2_cuda,signs_cuda, sizeOfSeq2, i);		
    	err = hipGetLastError();
    	if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
   	}	

	//copy signs_cuda to CPU memory
	stringSigns =(char*) malloc(sizeOfSeq2);
    	err = hipMemcpy(stringSigns,signs_cuda,sizeOfSeq2,hipMemcpyDeviceToHost);
 	if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
   	}
	tempBest.score = getAlignmentScore(stringSigns, sizeOfSeq2,w);
		if (tempBest.score > best.score) {
			best.score = tempBest.score;
			best.bestN = i;
			best.bestK = k;
		}
}
/* Loop End */

    // Free allocated memory on GPU
    if (hipFree(seq1_cuda) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(seq2_cuda) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(signs_cuda) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

return best;

}


